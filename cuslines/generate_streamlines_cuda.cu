#include "hip/hip_runtime.h"
/* Copyright (c) 2020, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 *    list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 *    this list of conditions and the following disclaimer in the documentation
 *    and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 *    contributors may be used to endorse or promote products derived from
 *    this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdio.h>
#include <stdlib.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <getopt.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <cfloat>
#include <omp.h>
#include <vector>
#include "cudamacro.h" /* for time() */
#include "globals.h"
//#include "utils.h"

#include <iostream>

#include "cuwsort.cuh"

#define MAX_NUM_DIR (128)

#define NTHR_GEN (128)

#define THR_X_BL (64)
#define THR_X_SL (32)

#define MAX_DIMS        (8)
#define MAX_STR_LEN     (256)

using namespace cuwsort;

//#define USE_FIXED_PERMUTATION
#ifdef USE_FIXED_PERMUTATION
//__device__ const int fixedPerm[] = {44, 47, 53,  0,  3,  3, 39,  9, 19, 21, 50, 36, 23,
//                                     6, 24, 24, 12,  1, 38, 39, 23, 46, 24, 17, 37, 25, 
//                                    13,  8,  9, 20, 51, 16, 51,  5, 15, 47,  0, 18, 35, 
//                                    24, 49, 51, 29, 19, 19, 14, 39, 32,  1,  9, 32, 31,
//                                    10, 52, 23};
__device__ const int fixedPerm[] = {
  47, 117,  67, 103,   9,  21,  36,  87,  70,  88, 140,  58,  39,  87,  88,  81,  25,  77,
  72,   9, 148, 115,  79,  82,  99,  29, 147, 147, 142,  32,   9, 127,  32,  31, 114,  28,
  34, 128, 128,  53, 133,  38,  17,  79, 132, 105,  42,  31, 120,   1,  65,  57,  35, 102,
 119,  11,  82,  91, 128, 142,  99,  53, 140, 121,  84,  68,   6,  47, 127, 131, 100,  78,
 143, 148,  23, 141, 117,  85,  48,  49,  69,  95,  94,   0, 113,  36,  48,  93, 131,  98,
  42, 112, 149, 127,   0, 138, 114,  43, 127,  23, 130, 121,  98,  62, 123,  82, 148,  50,
  14,  41,  58,  36,  10,  86,  43, 104,  11,   2,  51,  80,  32, 128,  38,  19,  42, 115,
  77,  30,  24, 125,   2,   3,  94, 107,  13, 112,  40,  72,  19,  95,  72,  67,  61,  14,
  96,   4, 139,  86, 121, 109};
#endif


template<int BDIM_X,
         typename REAL_T,
         typename REAL3_T>
__device__ int trilinear_interp_d(const int dimx,
                                  const int dimy,
                                  const int dimz,
                                  const int dimt,
                                  const REAL_T *__restrict__ dataf,
                                  const REAL3_T point,
                                        REAL_T *__restrict__ __vox_data) {

        const int tidx = threadIdx.x;
        //const int tidy = threadIdx.y;
#if 0        
        const int lid = (threadIdx.y*BDIM_X + threadIdx.x) % 32;
        const unsigned int WMASK = ((1ull << BDIM_X)-1) << (lid & (~(BDIM_X-1)));
#endif
        const REAL_T HALF = static_cast<REAL_T>(0.5);

        // all thr compute the same here
        if (point.x < -HALF || point.x+HALF >= dimx ||
            point.y < -HALF || point.y+HALF >= dimy ||
               point.z < -HALF || point.z+HALF >= dimz) {
                return -1;
        }

        long long  coo[3][2];
        REAL wgh[3][2]; // could use just one...

        const REAL_T ONE  = static_cast<REAL_T>(1.0);

        const REAL3_T fl = MAKE_REAL3(FLOOR(point.x),
                                      FLOOR(point.y),
                                      FLOOR(point.z));

        wgh[0][1] = point.x - fl.x; 
        wgh[0][0] = ONE-wgh[0][1]; 
        coo[0][0] = MAX(0, fl.x);
        coo[0][1] = MIN(dimx-1, coo[0][0]+1);

        wgh[1][1] = point.y - fl.y; 
        wgh[1][0] = ONE-wgh[1][1]; 
        coo[1][0] = MAX(0, fl.y);
        coo[1][1] = MIN(dimy-1, coo[1][0]+1);

        wgh[2][1] = point.z - fl.z; 
        wgh[2][0] = ONE-wgh[2][1]; 
        coo[2][0] = MAX(0, fl.z);
        coo[2][1] = MIN(dimz-1, coo[2][0]+1);

        //#pragma unroll
        for(int t = tidx; t < dimt; t += BDIM_X) {

                REAL_T __tmp = 0;

                #pragma unroll
                for(int i = 0; i < 2; i++) {
                        #pragma unroll
                        for(int j = 0; j < 2; j++) {
                                #pragma unroll
                                for(int k = 0; k < 2; k++) {
                                        __tmp += wgh[0][i]*wgh[1][j]*wgh[2][k]*
                                                 dataf[coo[0][i]*dimy*dimz*dimt +
                                                       coo[1][j]*dimz*dimt +
                                                       coo[2][k]*dimt +
                                                       t];
                                        /*
                                        if (tidx == 0 && threadIdx.y == 0 && t==0) {
                                                printf("wgh[0][%d]: %f, wgh[1][%d]: %f, wgh[2][%d]: %f\n",
                                                        i, wgh[0][i], j, wgh[1][j], k, wgh[2][k]);
                                                printf("dataf[%d][%d][%d][%d]: %f\n", coo[0][i], coo[1][j], coo[2][k], t+tidx,
                                                                dataf[coo[0][i]*dimy*dimz*dimt +
                                                                coo[1][j]*dimz*dimt +
                                                                coo[2][k]*dimt +
                                                                t+tidx]);
                                        }
                                        */
                                }
                        }
                }
                __vox_data[t] = __tmp;
        }
#if 0
        __syncwarp(WMASK);
        if (tidx == 0 && threadIdx.y == 0) {
                printf("point: %f, %f, %f\n", point.x, point.y, point.z);
                for(int i = 0; i < dimt; i++) {
                        printf("__vox_data[%d]: %f\n", i, __vox_data[i]);
                }
        }
#endif
        return 0;
}

template<int BDIM_X,
         typename VAL_T>
__device__ void ndotp_d(const int N,
			const int M,
			const VAL_T *__restrict__ srcV,
                        const VAL_T *__restrict__ srcM,
                              VAL_T *__restrict__ dstV) {

        const int tidx = threadIdx.x;

        const int lid = (threadIdx.y*BDIM_X + threadIdx.x) % 32;
        const unsigned int WMASK = ((1ull << BDIM_X)-1) << (lid & (~(BDIM_X-1)));

        //#pragma unroll
        for(int i = 0; i < N; i++) {

                VAL_T __tmp = 0;

                //#pragma unroll
                for(int j = 0; j < M; j += BDIM_X) {
                        if (j+tidx < M) {
                                __tmp += srcV[j+tidx]*srcM[i*M + j+tidx];
                        }
                }
                #pragma unroll
                for(int j = BDIM_X/2; j; j /= 2) {
#if 0
                        __tmp += __shfl_xor_sync(WMASK, __tmp, j, BDIM_X);
#else
                        __tmp += __shfl_down_sync(WMASK, __tmp, j, BDIM_X);
#endif
                }
                // values could be held by BDIM_X threads and written
                // together every BDIM_X iterations...

                if (tidx == 0) {
                        dstV[i] = __tmp;
                }
        }
        return;
}


template<int BDIM_X,
         typename VAL_T>
__device__ void ndotp_log_opdt_d(const int N,
			    const int M,
			    const VAL_T *__restrict__ srcV,
                            const VAL_T *__restrict__ srcM,
                                  VAL_T *__restrict__ dstV) {

        const int tidx = threadIdx.x;

        const int lid = (threadIdx.y*BDIM_X + threadIdx.x) % 32;
         const unsigned int WMASK = ((1ull << BDIM_X)-1) << (lid & (~(BDIM_X-1)));

        const VAL_T ONEP5 = static_cast<VAL_T>(1.5);

        //#pragma unroll
        for(int i = 0; i < N; i++) {

                VAL_T __tmp = 0;

                //#pragma unroll
                for(int j = 0; j < M; j += BDIM_X) {
                        if (j+tidx < M) {
                                const VAL_T v = srcV[j+tidx];
                                __tmp += -LOG(v)*(ONEP5+LOG(v))*v * srcM[i*M + j+tidx];
                        }
                }
                #pragma unroll
                for(int j = BDIM_X/2; j; j /= 2) {
#if 0
                        __tmp += __shfl_xor_sync(WMASK, __tmp, j, BDIM_X);
#else
                        __tmp += __shfl_down_sync(WMASK, __tmp, j, BDIM_X);
#endif
                }
                // values could be held by BDIM_X threads and written
                // together every BDIM_X iterations...

                if (tidx == 0) {
                        dstV[i] = __tmp;
                }
        }
        return;
}

template<int BDIM_X,
	 typename VAL_T>
__device__ void ndotp_log_csa_d(const int N,
				const int M,
				const VAL_T *__restrict__ srcV,
				const VAL_T *__restrict__ srcM,
				VAL_T *__restrict__ dstV) {

	const int tidx = threadIdx.x;

	const int lid = (threadIdx.y*BDIM_X + threadIdx.x) % 32;
	const unsigned int WMASK = ((1ull << BDIM_X)-1) << (lid & (~(BDIM_X-1)));
	// Clamp values
	constexpr VAL_T min = .001;
	constexpr VAL_T max = .999;

	//#pragma unroll
	for(int i = 0; i < N; i++) {

		VAL_T __tmp = 0;

		//#pragma unroll
		for(int j = 0; j < M; j += BDIM_X) {
			if (j+tidx < M) {
				const VAL_T v = MIN(MAX(srcV[j+tidx], min), max);
				__tmp += LOG(-LOG(v)) * srcM[i*M + j+tidx];
			}
		}
		#pragma unroll
		for(int j = BDIM_X/2; j; j /= 2) {
#if 0
			__tmp += __shfl_xor_sync(WMASK, __tmp, j, BDIM_X);
#else
			__tmp += __shfl_down_sync(WMASK, __tmp, j, BDIM_X);
#endif
		}
		// values could be held by BDIM_X threads and written
		// together every BDIM_X iterations...

		if (tidx == 0) {
			dstV[i] = __tmp;
		}
	}
	return;
}


template<int BDIM_X,
         typename REAL_T>
__device__ void fit_opdt(const int delta_nr,
                         const int hr_side,
                         const REAL_T *__restrict__ delta_q,
                         const REAL_T *__restrict__ delta_b,
                         const REAL_T *__restrict__ __msk_data_sh,
                         REAL_T *__restrict__ __h_sh,
                         REAL_T *__restrict__ __r_sh) {
        const int tidx = threadIdx.x;
        const int lid = (threadIdx.y*BDIM_X + threadIdx.x) % 32;
        const unsigned int WMASK = ((1ull << BDIM_X)-1) << (lid & (~(BDIM_X-1)));

        ndotp_log_opdt_d<BDIM_X>(delta_nr, hr_side, __msk_data_sh, delta_q, __r_sh);
        ndotp_d         <BDIM_X>(delta_nr, hr_side, __msk_data_sh, delta_b, __h_sh);
        __syncwarp(WMASK);
        #pragma unroll
        for(int j = tidx; j < delta_nr; j += BDIM_X) {
                __r_sh[j] -= __h_sh[j];
        }
        __syncwarp(WMASK);
}

template<int BDIM_X, typename REAL_T>
__device__ void fit_csa(const int delta_nr,
                        const int hr_side,
                        const REAL_T *__restrict__ fit_matrix,
                        const REAL_T *__restrict__ __msk_data_sh,
                        REAL_T *__restrict__ __r_sh) {
        const int tidx = threadIdx.x;
        const int lid = (threadIdx.y*BDIM_X + threadIdx.x) % 32;
        const unsigned int WMASK = ((1ull << BDIM_X)-1) << (lid & (~(BDIM_X-1)));

        constexpr REAL _n0_const = 0.28209479177387814; // .5 / sqrt(pi)
        ndotp_log_csa_d<BDIM_X>(delta_nr, hr_side, __msk_data_sh, fit_matrix, __r_sh);
        __syncwarp(WMASK);
        if (tidx == 0) {
                __r_sh[0] = _n0_const;
        }
        __syncwarp(WMASK);
}

// Based on dipy implementation:
// https://github.com/dipy/dipy/blob/104345de6dcbfd6627552b00d180e53db6c17381/dipy/reconst/csdeconv.py#L573C5-L573C13
// template<int BDIM_X, typename REAL_T>
// __device__ void fit_csd(const int delta_nr,
//                         const int hr_side,
//                         const REAL_T *__restrict__ __X,
//                         const REAL_T *__restrict__ B_reg,
//                         const REAL_T *__restrict__ __msk_data_sh,
//                         REAL_T *__restrict__ __r_sh) {
//         constexpr REAL_T mu = 1e-5;
//         constexpr REAL_T tau = 0.1;
//         REAL* __P, __z; // TODO: find somewhere to alloc these, or using existing space
//         nTmuln_d<BDIM_X>(delta_nr, hr_side, __X, __P); // TODO: P can be calculated beforehand
//         ndotp_d <BDIM_X>(delta_nr, hr_side, __X, __msk_data_sh, __z);

//         // TODO: finish this

// }

template<int BDIM_X, typename REAL_T>
__device__ void fit_model_coef(const ModelType model_type,
                               const int delta_nr, // delta_nr is number of ODF directions
                               const int hr_side, // hr_side is number of data directions
                               const REAL_T *__restrict__ delta_q,
                               const REAL_T *__restrict__ delta_b, // these are fit matrices the model can use, different for each model
                               const REAL_T *__restrict__ __msk_data_sh, // __msk_data_sh is the part of the data currently being operated on by this block
                               REAL_T *__restrict__ __h_sh, // these last two are modifications to the coefficients that will be returned
                               REAL_T *__restrict__ __r_sh) {
        switch(model_type) {
                case OPDT:
                        fit_opdt<BDIM_X>(delta_nr, hr_side, delta_q, delta_b, __msk_data_sh, __h_sh, __r_sh);
                        break;
                case CSA:
                        fit_csa<BDIM_X>(delta_nr, hr_side, delta_q, __msk_data_sh, __r_sh);
                        break;
                default:
                        printf("FATAL: Invalid Model Type.\n");
                        break;
        }
}

template<int BDIM_X,
         typename LEN_T,
         typename VAL_T>
__device__ VAL_T max_mask_transl_d(const int n,
				   const LEN_T *__restrict__ srcMsk,
                                   const VAL_T *__restrict__ srcVal,
                                   const VAL_T offset,
                                   const VAL_T minVal) {

        const int tidx = threadIdx.x;

        const int lid = (threadIdx.y*BDIM_X + threadIdx.x) % 32;
        const unsigned int WMASK = ((1ull << BDIM_X)-1) << (lid & (~(BDIM_X-1)));

        VAL_T __m = minVal;

        for(int i = tidx; i < n; i += BDIM_X) {
		const LEN_T sel = srcMsk[i];
		if (sel > 0) {
			__m = MAX(__m, srcVal[i]+offset);
		}
        }

        #pragma unroll
        for(int i = BDIM_X/2; i; i /= 2) {
                const VAL_T __tmp = __shfl_xor_sync(WMASK, __m, i, BDIM_X);
                __m = MAX(__m, __tmp);
        }

        return __m;
}

template<int BDIM_X,
         typename VAL_T>
__device__ VAL_T max_d(const int n, const VAL_T *__restrict__ src, const VAL_T minVal) {

        const int tidx = threadIdx.x;

        const int lid = (threadIdx.y*BDIM_X + threadIdx.x) % 32;
        const unsigned int WMASK = ((1ull << BDIM_X)-1) << (lid & (~(BDIM_X-1)));

        VAL_T __m = minVal;

        for(int i = tidx; i < n; i += BDIM_X) {
		__m = MAX(__m, src[i]);
        }

        #pragma unroll
        for(int i = BDIM_X/2; i; i /= 2) {
                const VAL_T __tmp = __shfl_xor_sync(WMASK, __m, i, BDIM_X);
                __m = MAX(__m, __tmp);
        }

        return __m;
}

template<int BDIM_X,
         typename VAL_T>
__device__ VAL_T min_d(const int n, const VAL_T *__restrict__ src, const VAL_T maxVal) {

        const int tidx = threadIdx.x;

        const int lid = (threadIdx.y*BDIM_X + threadIdx.x) % 32;
        const unsigned int WMASK = ((1ull << BDIM_X)-1) << (lid & (~(BDIM_X-1)));

        VAL_T __m = maxVal;

        for(int i = tidx; i < n; i += BDIM_X) {
		__m = MIN(__m, src[i]);
        }

        #pragma unroll
        for(int i = BDIM_X/2; i; i /= 2) {
                const VAL_T __tmp = __shfl_xor_sync(WMASK, __m, i, BDIM_X);
                __m = MIN(__m, __tmp);
        }

        return __m;
}
			
template<int BDIM_X,
         typename VAL_T>
__device__ VAL_T avgMask(const int mskLen,
			 const int *__restrict__ mask,
			 const VAL_T *__restrict__ data) {
        
	const int tidx = threadIdx.x;
        const int lid = (threadIdx.y*BDIM_X + threadIdx.x) % 32;

        const unsigned int WMASK = ((1ull << BDIM_X)-1) << (lid & (~(BDIM_X-1)));

        int   __myCnt = 0;
        VAL_T __mySum = 0;

        for(int i = tidx; i < mskLen; i += BDIM_X) {
		if(mask[i]) {
			__myCnt++;
			__mySum += data[i];
		}
        }

        #pragma unroll
        for(int i = BDIM_X/2; i; i /= 2) {
                __mySum += __shfl_xor_sync(WMASK, __mySum, i, BDIM_X);
                __myCnt += __shfl_xor_sync(WMASK, __myCnt, i, BDIM_X);
        }

        return __mySum/__myCnt;

}

template<int BDIM_X,
         int BDIM_Y,
         typename REAL_T,
         typename REAL3_T>
__device__ int peak_directions_d(const REAL_T  *__restrict__ odf,
                                       REAL3_T *__restrict__ dirs,
                                 const REAL3_T *__restrict__ sphere_vertices,
                                 const int2 *__restrict__ sphere_edges,
                                 const int num_edges,
				 int samplm_nr,
				 int *__restrict__ __shInd,
				 const REAL_T relative_peak_thres,
				 const REAL_T min_separation_angle) {

        const int tidx = threadIdx.x;

        const int lid = (threadIdx.y*BDIM_X + threadIdx.x) % 32;
        const unsigned int WMASK = ((1ull << BDIM_X)-1) << (lid & (~(BDIM_X-1)));

        const unsigned int lmask = (1 << lid)-1;

//        __shared__ int __shInd[BDIM_Y][SAMPLM_NR];

        #pragma unroll
        for(int j = tidx; j < samplm_nr; j += BDIM_X) {
		__shInd[j] = 0;
        }

        REAL_T odf_min = min_d<BDIM_X>(samplm_nr, odf, REAL_MAX);
        odf_min = MAX(0, odf_min);

        __syncwarp(WMASK);

        // local_maxima() + _compare_neighbors()
        // selecting only the indices corrisponding to maxima Ms
        // such that M-odf_min >= relative_peak_thres
        //#pragma unroll
        for(int j = 0; j < num_edges; j += BDIM_X) {
                if (j+tidx < num_edges) {
                        const int u_ind = sphere_edges[j+tidx].x;
                        const int v_ind = sphere_edges[j+tidx].y;

                        //if (u_ind >= NUM_EDGES || v_ind >= NUM_EDGES) { ERROR; }

                        const REAL_T u_val = odf[u_ind];
                        const REAL_T v_val = odf[v_ind];

                        //if (u_val != u_val || v_val != v_val) { ERROR_NANs; }

                        // only check that they are not equal
                        //if (u_val != v_val) {
                        //        __shInd[tidy][u_val < v_val ? u_ind : v_ind] = -1; // benign race conditions...
                        //}
                        if (u_val < v_val) {
                                atomicExch(__shInd+u_ind, -1);
                                atomicOr(  __shInd+v_ind,  1);
                        } else if (v_val < u_val) {
                                atomicExch(__shInd+v_ind, -1);
                                atomicOr(  __shInd+u_ind,  1);
                        }
                }
        }
        __syncwarp(WMASK);

        const REAL_T compThres = relative_peak_thres*max_mask_transl_d<BDIM_X>(samplm_nr, __shInd, odf, -odf_min, REAL_MIN);
#if 1
/*
        if (!tidy && !tidx) {
                for(int j = 0; j < SAMPLM_NR; j++) {
                        printf("local_max[%d]: %d (%f)\n", j, __shInd[tidy][j], odf[j]);
                }
                printf("maxMax with offset %f: %f\n", -odf_min, compThres);
        }
        __syncwarp(WMASK);
*/
        // compact indices of positive values to the right
        int n = 0;

        for(int j = 0; j < samplm_nr; j += BDIM_X) {

                const int __v = (j+tidx < samplm_nr) ? __shInd[j+tidx] : -1;
                const int __keep = (__v > 0) && ((odf[j+tidx]-odf_min) >= compThres);
                const int __msk = __ballot_sync(WMASK, __keep);

//__syncwarp(WMASK); // unnecessary
                if (__keep) {
                        const int myoff = __popc(__msk & lmask);
                        __shInd[n + myoff] = j+tidx;
                }
                n += __popc(__msk);
//__syncwarp(WMASK); // should be unnecessary
        }
        __syncwarp(WMASK);
/*
        if (!tidy && !tidx) {
                for(int j = 0; j < n; j++) {
                        printf("local_max_compact[%d]: %d\n", j, __shInd[tidy][j]);
                }
        }
        __syncwarp(WMASK);
*/

        // sort local maxima indices
        if (n < BDIM_X) {
                REAL_T k = REAL_MIN;
                int    v = 0;
                if (tidx < n) {
                        v = __shInd[tidx];
                        k = odf[v];
                }
                warp_sort<32, BDIM_X, WSORT_DIR_DEC>(&k, &v);
                __syncwarp(WMASK);

                if (tidx < n) {
                        __shInd[tidx] = v;
                }
        } else {
                // ERROR !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
        }
        __syncwarp(WMASK);

        // __shInd[tidy][] contains the indices in odf correspoding to
        // normalized maxima NOT sorted!
        if (n != 0) {
                // remove_similar_vertices()
                // PRELIMINARY INEFFICIENT, SINGLE TH, IMPLEMENTATION
                if (tidx == 0) {
                        const REAL_T cos_similarity = COS(min_separation_angle);

                        dirs[0] = sphere_vertices[__shInd[0]];

                        int k = 1;
                        for(int i = 1; i < n; i++) {

                                const REAL3_T abc = sphere_vertices[__shInd[i]];

                                int j = 0;
                                for(; j < k; j++) {
                                        const REAL_T cos = FABS(abc.x*dirs[j].x+
                                                                abc.y*dirs[j].y+
                                                                abc.z*dirs[j].z);
                                        if (cos > cos_similarity) {
                                                break;
                                        }
                                }
                                if (j == k) {
                                        dirs[k++] = abc;
                                }
                        }
                        n = k;
                }
                n = __shfl_sync(WMASK, n, 0, BDIM_X);
                __syncwarp(WMASK);

        }
/*
        if (!tidy && !tidx) {
                for(int j = 0; j < n; j++) {
                        printf("local_max_compact_uniq[%d]: %d\n", j, __shInd[tidy][j]);
                }
        }
        __syncwarp(WMASK);
*/
#else
        const int indMax = max_d<BDIM_X, SAMPLM_NR>(__shInd[tidy], -1);
        if (indMax != -1) {
                __ret = MAKE_REAL3(sphere_vertices[indMax][0],
                                   sphere_vertices[indMax][1],
                                   sphere_vertices[indMax][2]);
        }
#endif
        return n;
}

template<int BDIM_X,
         int BDIM_Y,
         typename REAL_T,
         typename REAL3_T>
__device__ int closest_peak_d(const REAL_T max_angle,
			      const REAL3_T  direction, //dir
                              const int npeaks,
                              const REAL3_T *__restrict__ peaks,
                                    REAL3_T *__restrict__ peak) {// dirs,

        const int tidx = threadIdx.x;

        const int lid = (threadIdx.y*BDIM_X + threadIdx.x) % 32;
        const unsigned int WMASK = ((1ull << BDIM_X)-1) << (lid & (~(BDIM_X-1)));

        //const REAL_T cos_similarity = COS(MAX_ANGLE_P);
        const REAL_T cos_similarity = COS(max_angle);
#if 0
        if (!threadIdx.y && !tidx) {
                printf("direction: (%f, %f, %f)\n",
                        direction.x, direction.y, direction.z);
        }
        __syncwarp(WMASK);
#endif
        REAL_T cpeak_dot = 0;
        int    cpeak_idx = -1;
        for(int j = 0; j < npeaks; j += BDIM_X) {
                if (j+tidx < npeaks) {
#if 0
                        if (!threadIdx.y && !tidx) {
                                printf("j+tidx: %d, peaks[j+tidx]: (%f, %f, %f)\n",
                                        j+tidx, peaks[j+tidx].x, peaks[j+tidx].y, peaks[j+tidx].z);
                        }
#endif
                        const REAL_T dot = direction.x*peaks[j+tidx].x+
                                           direction.y*peaks[j+tidx].y+
                                           direction.z*peaks[j+tidx].z;

                        if (FABS(dot) > FABS(cpeak_dot)) {
                                cpeak_dot = dot;
                                cpeak_idx = j+tidx;
                        }
                }
        }
#if 0
        if (!threadIdx.y && !tidx) {
                printf("cpeak_idx: %d, cpeak_dot: %f\n", cpeak_idx, cpeak_dot);
        }
        __syncwarp(WMASK);
#endif

        #pragma unroll
        for(int j = BDIM_X/2; j; j /= 2) {

                const REAL_T dot = __shfl_xor_sync(WMASK, cpeak_dot, j, BDIM_X);
                const int    idx = __shfl_xor_sync(WMASK, cpeak_idx, j, BDIM_X);
                if (FABS(dot) > FABS(cpeak_dot)) {
                        cpeak_dot = dot;
                        cpeak_idx = idx;
                }
        }
#if 0
        if (!threadIdx.y && !tidx) {
                printf("cpeak_idx: %d, cpeak_dot: %f, cos_similarity: %f\n", cpeak_idx, cpeak_dot, cos_similarity);
        }
        __syncwarp(WMASK);
#endif
        if (cpeak_idx >= 0) {
                if (cpeak_dot >= cos_similarity) {
                        peak[0] = peaks[cpeak_idx];
                        return 1;
                }
                if (cpeak_dot <= -cos_similarity) {
                        peak[0] = MAKE_REAL3(-peaks[cpeak_idx].x,
                                             -peaks[cpeak_idx].y,
                                             -peaks[cpeak_idx].z);
                        return 1;
                }
        }
        return 0;
}

template<int BDIM_X,
	 typename LEN_T,
	 typename MSK_T,
	 typename VAL_T>
__device__ LEN_T maskGet(const LEN_T n, 
			 const MSK_T *__restrict__ mask,
			 const VAL_T *__restrict__ plain,
			       VAL_T *__restrict__ masked) {

	const int tidx = threadIdx.x;
	
        const int lid = (threadIdx.y*BDIM_X + threadIdx.x) % 32;
        const unsigned int WMASK = ((1ull << BDIM_X)-1) << (lid & (~(BDIM_X-1)));

	const int __laneMask = (1 << tidx)-1;

	int woff = 0;
	for(int j = 0; j < n; j += BDIM_X) {

		const int __act = (j+tidx < n) ? !mask[j+tidx] : 0;
		const int __msk = __ballot_sync(WMASK, __act);

		const int toff = __popc(__msk & __laneMask);
		if (__act) {
			masked[woff+toff] = plain[j+tidx];
		}
		woff += __popc(__msk);
	}
	return woff;
}

template<int BDIM_X,
	 typename LEN_T,
	 typename MSK_T,
	 typename VAL_T>
__device__ void maskPut(const LEN_T n, 
			const MSK_T *__restrict__ mask,
			const VAL_T *__restrict__ masked,
			      VAL_T *__restrict__ plain) {

	const int tidx = threadIdx.x;
	
        const int lid = (threadIdx.y*BDIM_X + threadIdx.x) % 32;
        const unsigned int WMASK = ((1ull << BDIM_X)-1) << (lid & (~(BDIM_X-1)));

	const int __laneMask = (1 << tidx)-1;

	int woff = 0;
	for(int j = 0; j < n; j += BDIM_X) {

		const int __act = (j+tidx < n) ? !mask[j+tidx] : 0;
		const int __msk = __ballot_sync(WMASK, __act);

		const int toff = __popc(__msk & __laneMask);
		if (__act) {
			plain[j+tidx] = masked[woff+toff];
		}
		woff += __popc(__msk);
	}
	return;
}

template<typename REAL_T>
__device__ void printArray(const char *name, int ncol, int n, REAL_T *arr) {
	if (!threadIdx.x && !threadIdx.y && !blockIdx.x) {
		printf("%s:\n", name);

		for(int j = 0; j < n; j++) {
			if ((j%ncol)==0) printf("\n");
			printf("%10.8f ", arr[j]);
		}
		printf("\n");
	}
}

template<int BDIM_X,
         int BDIM_Y,
         bool IS_START,
         typename REAL_T,
         typename REAL3_T>
__device__ int get_direction_prob_d(hiprandStatePhilox4_32_10_t *st,
                                    const REAL_T *__restrict__ pmf,
                                    const REAL_T max_angle,
                                    const REAL_T relative_peak_thres,
                                    const REAL_T min_separation_angle,
                                    REAL3_T dir,
                                    const int dimx,
                                    const int dimy,
                                    const int dimz,
                                    const int dimt,
                                    const REAL3_T point,
                                    const int samplm_nr,
                                    const REAL3_T *__restrict__ sphere_vertices,
                                    const int2 *__restrict__ sphere_edges,
                                    const int num_edges,
                                    REAL3_T *__restrict__ dirs) {
        const int tidx = threadIdx.x;
        const int tidy = threadIdx.y;
	
        const int lid = (threadIdx.y*BDIM_X + threadIdx.x) % 32;
        const unsigned int WMASK = ((1ull << BDIM_X)-1) << (lid & (~(BDIM_X-1)));

	const int n32dimt = ((dimt+31)/32)*32;

	extern REAL_T __shared__ __sh[];

	REAL_T *__pmf_data_sh = reinterpret_cast<REAL_T *>(__sh);
	int *__shInd = reinterpret_cast<int *>(__pmf_data_sh) + BDIM_Y*n32dimt; // only used if IS_START is 1

	__pmf_data_sh += tidy*n32dimt;
	__shInd += tidy*n32dimt;

        // pmf = self.pmf_gen.get_pmf_c(&point[0], pmf)
        const int rv = trilinear_interp_d<BDIM_X>(dimx, dimy, dimz, dimt, pmf, point, __pmf_data_sh);
        if (rv != 0) {
                return 0;
        }

        // for i in range(_len):
        //     if pmf[i] > max_pmf:
        //         max_pmf = pmf[i]
        // absolute_pmf_threshold = pmf_threshold * max_pmf
        const REAL_T absolpmf_thresh = PMF_THRESHOLD_P * max_d<BDIM_X>(samplm_nr, __pmf_data_sh, REAL_MIN);
        __syncwarp(WMASK);

        // for i in range(_len):
        //     if pmf[i] < absolute_pmf_threshold:
        //         pmf[i] = 0.0
        for(int i = tidx; i < samplm_nr; i += BDIM_X) {
                if (__pmf_data_sh[i] < absolpmf_thresh) {
                        __pmf_data_sh[i] = 0.0;
                }
        }
        __syncwarp(WMASK);

        if (IS_START) {
                return peak_directions_d<BDIM_X,
                                         BDIM_Y>(__pmf_data_sh,
                                                 dirs,
                                                 sphere_vertices,
                                                 sphere_edges,
                                                 num_edges,
                                                 samplm_nr,
                                                 __shInd,
                                                 relative_peak_thres,
                                                 min_separation_angle);
        } else {
                REAL_T __tmp;
                #ifdef DEBUG
                        __syncwarp();
                        if (tidx == 0) {
                                printArray("__pmf_data_sh initial", 8, samplm_nr, __pmf_data_sh);
                                printf("absolpmf_thresh %10.8f\n", absolpmf_thresh);
                                printf("--->            dir %10.8f, %10.8f, %10.8f\n", dir.x, dir.y, dir.z);
                                printf("--->            point %10.8f, %10.8f, %10.8f\n", point.x, point.y, point.z);
                                if (sqrt(dir.x*dir.x + dir.y*dir.y + dir.z*dir.z) >= 1.05){
                                        printf("ERROR dir %10.8f, %10.8f, %10.8f\n", dir.x, dir.y, dir.z);
                                }
                        }
                #endif

                // // These should not be relevant
                // if norm(&direction[0]) == 0:
                //     return 1
                // normalize(&direction[0])

                // for i in range(_len):
                //         cos_sim = self.vertices[i][0] * direction[0] \
                //                 + self.vertices[i][1] * direction[1] \
                //                 + self.vertices[i][2] * direction[2]
                //         if cos_sim < 0:
                //                 cos_sim = cos_sim * -1
                //         if cos_sim < self.cos_similarity:
                //                 pmf[i] = 0
                const REAL_T cos_similarity = COS(max_angle);

                for(int i = tidx; i < samplm_nr; i += BDIM_X) {
                        const REAL_T dot = dir.x*sphere_vertices[i].x+
                                           dir.y*sphere_vertices[i].y+
                                           dir.z*sphere_vertices[i].z;

                        if (FABS(dot) < cos_similarity) {
                                __pmf_data_sh[i] = 0.0;
                        }
                }
                __syncwarp(WMASK);

                #ifdef DEBUG
                        __syncwarp();
                        if (tidx == 0) {
                                printArray("__pmf_data_sh after filtering", 8, samplm_nr, __pmf_data_sh);
                        }
                #endif

                // cumsum(pmf, pmf, _len)
                for (int j = 0; j < samplm_nr; j += BDIM_X) {
                        if ((tidx == 0) && (j != 0)) {
                                __pmf_data_sh[j] += __pmf_data_sh[j-1];
                        }
                        __syncwarp(WMASK);

                        #pragma unroll
                        for(int i = 1; i < BDIM_X; i *= 2) {
                                if ((tidx >= i) && (j+tidx < samplm_nr)) {
                                        __tmp = __pmf_data_sh[j+tidx-i];
                                }
                                __syncwarp(WMASK);
                                if ((tidx >= i) && (j+tidx < samplm_nr)) {
                                        __pmf_data_sh[j+tidx] += __tmp;
                                }
                                __syncwarp(WMASK);
                        }
                }

                #ifdef DEBUG
                        __syncwarp();
                        if (tidx == 0) {
                                printArray("__pmf_data_sh after cumsum", 8, samplm_nr, __pmf_data_sh);
                        }
                #endif

                // last_cdf = pmf[_len - 1]
                // if last_cdf == 0:
                //         return 1
                if (tidx == 0) {
                        __tmp = __pmf_data_sh[samplm_nr - 1];
                }
                REAL_T last_cdf = __shfl_sync(WMASK, __tmp, 0, BDIM_X);
                if (last_cdf == 0) {
                        return 0;
                }

                // idx = where_to_insert(pmf, random() * last_cdf, _len)
                if (tidx == 0) {
                        __tmp = hiprand_uniform(st) * last_cdf;
                }
                REAL_T selected_cdf = __shfl_sync(WMASK, __tmp, 0, BDIM_X);
                int indProb = samplm_nr;
                for(int i = samplm_nr - 1 - tidx; i >= 0; i-= BDIM_X) {
                        if (selected_cdf >= __pmf_data_sh[i]) {
                                if ((i < samplm_nr) && (selected_cdf < __pmf_data_sh[i + 1])) {
                                        indProb = i + 1;
                                } else {
                                        indProb = MIN(samplm_nr - 1, i + BDIM_X);
                                }
                                break;
                        }
                }

                #pragma unroll
                for(int i = BDIM_X/2; i; i /= 2) {
                        __tmp = __shfl_xor_sync(WMASK, indProb, i, BDIM_X);
                        indProb = MIN(indProb, __tmp);
                }

                #ifdef DEBUG
                        __syncwarp();
                        if (tidx == 0) {
                                printf("last_cdf %10.8f\n", last_cdf);
                                printf("selected_cdf %10.8f\n", selected_cdf);
                                printf("indProb %i out of %i\n", indProb, samplm_nr);
                        }
                #endif

                // newdir = self.vertices[idx]
                // if (direction[0] * newdir[0]
                //     + direction[1] * newdir[1]
                //     + direction[2] * newdir[2] > 0):
                //     copy_point(&newdir[0], &direction[0])
                // else:
                //     newdir[0] = newdir[0] * -1
                //     newdir[1] = newdir[1] * -1
                //     newdir[2] = newdir[2] * -1
                //     copy_point(&newdir[0], &direction[0])
                if (tidx == 0) {
                        if ((dir.x * sphere_vertices[indProb].x +
                             dir.y * sphere_vertices[indProb].y +
                             dir.z * sphere_vertices[indProb].z) > 0) {
                                dirs[0] = MAKE_REAL3(sphere_vertices[indProb].x,
                                                     sphere_vertices[indProb].y,
                                                     sphere_vertices[indProb].z);
                        } else {
                                dirs[0] = MAKE_REAL3(-sphere_vertices[indProb].x,
                                                     -sphere_vertices[indProb].y,
                                                     -sphere_vertices[indProb].z);
                        }
                }

                #ifdef DEBUG
                        if (tidx == 0) {
                                if ((dirs[0].x == dir.x) && (dirs[0].y == dir.y) && (dirs[0].z == dir.z)) {
                                printf("ERROR dir %10.8f, %10.8f, %10.8f\n", dirs[0].x, dirs[0].y, dirs[0].z);
                                printf("last_cdf %10.8f\n", last_cdf);
                                printf("selected_cdf %10.8f\n", selected_cdf);
                                printf("indProb %i out of %i\n", indProb, samplm_nr);
                                }
                                if (sqrt(dirs[0].x*dirs[0].x + dirs[0].y*dirs[0].y + dirs[0].z*dirs[0].z) >= 1.1) {
                                        printf("ERROR dir %10.8f, %10.8f, %10.8f\n", dirs[0].x, dirs[0].y, dirs[0].z);
                                }
                        }
                #endif
                return 1;
        }
}

template<int BDIM_X,
         int BDIM_Y,
         int NATTEMPTS,
         typename REAL_T,
         typename REAL3_T>
__device__ int get_direction_boot_d(
                                hiprandStatePhilox4_32_10_t *st,
                                const ModelType model_type,
                                const REAL_T max_angle,
                                const REAL_T min_signal,
                                const REAL_T relative_peak_thres,
                                const REAL_T min_separation_angle,
                                REAL3_T dir,
                                const int dimx,
                                const int dimy,
                                const int dimz,
                                const int dimt,
                                const REAL_T *__restrict__ dataf,
                                const int *__restrict__ b0s_mask, // not using this (and its opposite, dwi_mask)
                                                                  // but not clear if it will never be needed so
                                                                  // we'll keep it here for now...
                                const REAL3_T point,
                                const REAL_T *__restrict__ H, 
                                const REAL_T *__restrict__ R,
                                // model unused
                                // max_angle, pmf_threshold from global defines
                                // b0s_mask already passed
                                // min_signal from global defines
                                const int delta_nr,
                                const REAL_T *__restrict__ delta_b,
                                const REAL_T *__restrict__ delta_q, // fit_matrix
                                const int samplm_nr,
                                const REAL_T *__restrict__ sampling_matrix,
                                const REAL3_T *__restrict__ sphere_vertices,
                                const int2 *__restrict__ sphere_edges,
                                const int num_edges,
                                REAL3_T *__restrict__ dirs) {

        const int tidx = threadIdx.x;
        const int tidy = threadIdx.y;
	
        const int lid = (threadIdx.y*BDIM_X + threadIdx.x) % 32;
        const unsigned int WMASK = ((1ull << BDIM_X)-1) << (lid & (~(BDIM_X-1)));

	const int n32dimt = ((dimt+31)/32)*32;

	extern REAL_T __shared__ __sh[];

	REAL_T *__vox_data_sh = reinterpret_cast<REAL_T *>(__sh);
	REAL_T *__msk_data_sh = __vox_data_sh + BDIM_Y*n32dimt;

	REAL_T *__r_sh = __msk_data_sh + BDIM_Y*n32dimt;
	REAL_T *__h_sh = __r_sh + BDIM_Y*MAX(n32dimt, samplm_nr);

	__vox_data_sh += tidy*n32dimt;
	__msk_data_sh += tidy*n32dimt;

	__r_sh += tidy*MAX(n32dimt, samplm_nr);
	__h_sh += tidy*MAX(n32dimt, samplm_nr);
	
	// compute hr_side (may be passed from python)
	int hr_side = 0;
	for(int j = tidx; j < dimt; j += BDIM_X) {
		hr_side += !b0s_mask[j] ? 1 : 0;
	}
        #pragma unroll
        for(int i = BDIM_X/2; i; i /= 2) {
                hr_side += __shfl_xor_sync(WMASK, hr_side, i, BDIM_X);
        }

        #pragma unroll
        for(int i = 0; i < NATTEMPTS; i++) {

                const int rv = trilinear_interp_d<BDIM_X>(dimx, dimy, dimz, dimt, dataf, point, __vox_data_sh);

		const int nmsk = maskGet<BDIM_X>(dimt, b0s_mask, __vox_data_sh, __msk_data_sh);

		//if (!tidx && !threadIdx.y && !blockIdx.x) {
		//
		//	printf("interp of %f, %f, %f\n", point.x, point.y, point.z);
		//	printf("hr_side: %d\n", hr_side);
		//	printArray("vox_data", 6, dimt, __vox_data_sh[tidy]);
		//	printArray("msk_data", 6, nmsk, __msk_data_sh[tidy]);
		//}
		//break;

                __syncwarp(WMASK);

                if (rv == 0) {

                        ndotp_d<BDIM_X>(hr_side, hr_side, __msk_data_sh, R, __r_sh);
			//__syncwarp();
			//printArray("__r", 5, hr_side*hr_side, R);
			//printArray("__r_sh", 6, hr_side, __r_sh[tidy]);

                        ndotp_d<BDIM_X>(hr_side, hr_side, __msk_data_sh, H, __h_sh);
			//__syncwarp();
			//printArray("__h_sh", 6, hr_side, __h_sh[tidy]);

                        __syncwarp(WMASK);

                        for(int j = 0; j < hr_side; j += BDIM_X) {
                                if (j+tidx < hr_side) {
#ifdef USE_FIXED_PERMUTATION
                                        const int srcPermInd = fixedPerm[j+tidx];
#else
                                        const int srcPermInd = hiprand(st) % hr_side;
//                                        if (srcPermInd < 0 || srcPermInd >= hr_side) {
//                                                printf("srcPermInd: %d\n", srcPermInd);
//                                        }
#endif
					__h_sh[j+tidx] += __r_sh[srcPermInd];
					//__h_sh[j+tidx] += __r_sh[j+tidx];
                                }
                        }
			__syncwarp(WMASK);

			//printArray("h+perm(r):", 6, hr_side, __h_sh[tidy]);
			//__syncwarp();
		
			// vox_data[dwi_mask] = masked_data
			maskPut<BDIM_X>(dimt, b0s_mask, __h_sh, __vox_data_sh);
			__syncwarp(WMASK);

			//printArray("vox_data[dwi_mask]:", 6, dimt, __vox_data_sh[tidy]);
			//__syncwarp();

			for(int j = tidx; j < dimt; j += BDIM_X) {
				//__vox_data_sh[j] = MAX(MIN_SIGNAL_P, __vox_data_sh[j]);
				__vox_data_sh[j] = MAX(min_signal, __vox_data_sh[j]);
			}
			__syncwarp(WMASK);

			const REAL_T denom = avgMask<BDIM_X>(dimt, b0s_mask, __vox_data_sh);

			for(int j = tidx; j < dimt; j += BDIM_X) {
				__vox_data_sh[j] /= denom;
			}
			__syncwarp();

			//if (!tidx && !threadIdx.y && !blockIdx.x) {
			//	printf("denom: %f\n", denom);
			//}
			////break;
			//if (!tidx && !threadIdx.y && !blockIdx.x) {
			//
			//	printf("__vox_data_sh:\n");
			//	printArray("vox_data", 6, dimt, __vox_data_sh[tidy]);
			//}
			//break;

			maskGet<BDIM_X>(dimt, b0s_mask, __vox_data_sh, __msk_data_sh);
			__syncwarp(WMASK);

                        fit_model_coef<BDIM_X>(model_type, delta_nr, hr_side, delta_q, delta_b, __msk_data_sh, __h_sh, __r_sh);

                        // __r_sh[tidy] <- python 'coef'

                        ndotp_d<BDIM_X>(samplm_nr, delta_nr, __r_sh, sampling_matrix, __h_sh);

                        // __h_sh[tidy] <- python 'pmf'
                } else {
                        #pragma unroll
                        for(int j = tidx; j < samplm_nr; j += BDIM_X) {
				__h_sh[j] = 0;
                        }
                        // __h_sh[tidy] <- python 'pmf'
                }
                __syncwarp(WMASK);
#if 0
                if (!threadIdx.y && threadIdx.x == 0) {
                        for(int j = 0; j < samplm_nr; j++) {
                                printf("pmf[%d]: %f\n", j, __h_sh[tidy][j]);
                        }
                }
                //return;
#endif
                const REAL_T abs_pmf_thr = PMF_THRESHOLD_P*max_d<BDIM_X>(samplm_nr, __h_sh, REAL_MIN);
                __syncwarp(WMASK);

                #pragma unroll
                for(int j = tidx; j < samplm_nr; j += BDIM_X) {
			const REAL_T __v = __h_sh[j];
			if (__v < abs_pmf_thr) {
				__h_sh[j] = 0;
			}
                }
                __syncwarp(WMASK);
#if 0
                if (!threadIdx.y && threadIdx.x == 0) {
                        printf("abs_pmf_thr: %f\n", abs_pmf_thr);
                        for(int j = 0; j < samplm_nr; j++) {
                                printf("pmfNORM[%d]: %f\n", j, __h_sh[tidy][j]);
                        }
                }
                //return;
#endif
#if 0
                if init:
                        directions = peak_directions(pmf, sphere)[0]
                        return directions
                else:
                        peaks = peak_directions(pmf, sphere)[0]
                        if (len(peaks) > 0):
                                return closest_peak(directions, peaks, cos_similarity)
#endif
                const int ndir = peak_directions_d<BDIM_X,
                                                   BDIM_Y>(__h_sh, dirs,
                                                           sphere_vertices,
                                                           sphere_edges,
                                                           num_edges,
							   samplm_nr,
							   reinterpret_cast<int *>(__r_sh), // reuse __r_sh as shInd in func which is large enough
							   relative_peak_thres,
							   min_separation_angle);
                if (NATTEMPTS == 1) { // init=True...
                        return ndir; // and dirs;
                } else { // init=False...
                        if (ndir > 0) {
                                /*
                                if (!threadIdx.y && threadIdx.x == 0 && ndir > 1) {
                                        printf("NATTEMPTS=5 and ndir: %d!!!\n", ndir);
                                }
                                */
                                REAL3_T peak;
                                const int foundPeak = closest_peak_d<BDIM_X, BDIM_Y, REAL_T, REAL3_T>(max_angle, dir, ndir, dirs, &peak);
                                __syncwarp(WMASK);
                                if (foundPeak) {
                                        if (tidx == 0) {
                                                dirs[0] = peak;
                                        }
                                        return 1;
                                }
                        }
                }
        }
        return 0;
}

enum {OUTSIDEIMAGE, INVALIDPOINT, TRACKPOINT, ENDPOINT};

template<int BDIM_X,
         int BDIM_Y,
         typename REAL_T,
         typename REAL3_T>
__device__ int check_point_d(const REAL_T tc_threshold,
			     const REAL3_T point,
                             const int dimx,
                             const int dimy,
                             const int dimz,
                             const REAL_T *__restrict__ metric_map) {

        const int tidy = threadIdx.y;

        const int lid = (threadIdx.y*BDIM_X + threadIdx.x) % 32;
        const unsigned int WMASK = ((1ull << BDIM_X)-1) << (lid & (~(BDIM_X-1)));

        __shared__ REAL_T __shInterpOut[BDIM_Y];

        const int rv = trilinear_interp_d<BDIM_X>(dimx, dimy, dimz, 1, metric_map, point, __shInterpOut+tidy);
        __syncwarp(WMASK);
#if 0
        if (threadIdx.y == 1 && threadIdx.x == 0) {
                printf("__shInterpOut[tidy]: %f, TC_THRESHOLD_P: %f\n", __shInterpOut[tidy], TC_THRESHOLD_P);
        }
#endif
        if (rv != 0) {
                return OUTSIDEIMAGE;
        }
        //return (__shInterpOut[tidy] > TC_THRESHOLD_P) ? TRACKPOINT : ENDPOINT;
        return (__shInterpOut[tidy] > tc_threshold) ? TRACKPOINT : ENDPOINT;
}

template<int BDIM_X,
         int BDIM_Y,
         typename REAL_T,
         typename REAL3_T>
__device__ int tracker_d(hiprandStatePhilox4_32_10_t *st,
			 const ModelType model_type,
			 const REAL_T max_angle,
			 const REAL_T min_signal,
			 const REAL_T tc_threshold,
			 const REAL_T step_size,
			 const REAL_T relative_peak_thres,
			 const REAL_T min_separation_angle,
                         REAL3_T seed,
                         REAL3_T first_step,
                         REAL3_T voxel_size,
                         const int dimx,
                         const int dimy,
                         const int dimz,
                         const int dimt,
                         const REAL_T *__restrict__ dataf,
                         const int *__restrict__ b0s_mask, // not using this (and its opposite, dwi_mask)
                         const REAL_T *__restrict__ H, 
                         const REAL_T *__restrict__ R,
                         // model unused
                         // step_size from global defines
                         // max_angle, pmf_threshold from global defines
                         // b0s_mask already passed
                         // min_signal from global defines
                         // tc_threshold from global defines
                         // pmf_threashold from global defines
                         const REAL_T *__restrict__ metric_map,
			 const int delta_nr,
                         const REAL_T *__restrict__ delta_b,
                         const REAL_T *__restrict__ delta_q, // fit_matrix
			 const int samplm_nr,
                         const REAL_T *__restrict__ sampling_matrix,
                         const REAL3_T *__restrict__ sphere_vertices,
                         const int2 *__restrict__ sphere_edges,
                         const int num_edges,
                               REAL3_T *__restrict__ __shDir,
                               int *__restrict__ nsteps,
                               REAL3_T *__restrict__ streamline) {

        const int tidx = threadIdx.x;
        //const int tidy = threadIdx.y;

        const int lid = (threadIdx.y*BDIM_X + threadIdx.x) % 32;
        const unsigned int WMASK = ((1ull << BDIM_X)-1) << (lid & (~(BDIM_X-1)));

        int tissue_class = TRACKPOINT;

        REAL3_T point = seed;
        REAL3_T direction = first_step;

        if (tidx == 0) {
                streamline[0] = point;
#if 0
                if (threadIdx.y == 1) {
                        printf("streamline[0]: %f, %f, %f\n", point.x, point.y, point.z);
                }
#endif
        }
        __syncwarp(WMASK);

        int i;
        for(i = 1; i < MAX_SLINE_LEN; i++) {
                int ndir;
                if (model_type == PROB) {
                        ndir = get_direction_prob_d<BDIM_X,
                                                    BDIM_Y,
                                                    0>(
                                                        st,
                                                        dataf,
                                                        max_angle,
                                                        relative_peak_thres,
                                                        min_separation_angle,
                                                        direction,
                                                        dimx, dimy, dimz, dimt,
                                                        point,
                                                        samplm_nr,
                                                        sphere_vertices,
                                                        sphere_edges,
                                                        num_edges,
                                                        __shDir);
                } else {
                        // call get_direction_boot_d() with NATTEMPTS=5
                        ndir = get_direction_boot_d<BDIM_X,
                                                    BDIM_Y,
                                                    5>(
                                                        st,
                                                        model_type,
                                                        max_angle,
                                                        min_signal,
                                                        relative_peak_thres,
                                                        min_separation_angle,
                                                        direction,
                                                        dimx, dimy, dimz, dimt, dataf,
                                                        b0s_mask /* !dwi_mask */,
                                                        point,
                                                        H, R,
                                                        // model unused
                                                        // max_angle, pmf_threshold from global defines
                                                        // b0s_mask already passed
                                                        // min_signal from global defines
                                                        delta_nr,
                                                        delta_b, delta_q, // fit_matrix
                                                        samplm_nr,
                                                        sampling_matrix,
                                                        sphere_vertices,
                                                        sphere_edges,
                                                        num_edges,
                                                        __shDir);
                }
                __syncwarp(WMASK);
                direction = __shDir[0];
                __syncwarp(WMASK);

                if (ndir == 0) {
                        break;
                }
#if 0
                if (threadIdx.y == 1 && threadIdx.x == 0) {
                        printf("tracker: i: %d, direction: (%f, %f, %f)\n", i, direction.x, direction.y, direction.z);
                }
                //return;
#endif
                //point.x += (direction.x / voxel_size.x) * STEP_SIZE_P;
                //point.y += (direction.y / voxel_size.y) * STEP_SIZE_P;
                //point.z += (direction.z / voxel_size.z) * STEP_SIZE_P;
                point.x += (direction.x / voxel_size.x) * step_size;
                point.y += (direction.y / voxel_size.y) * step_size;
                point.z += (direction.z / voxel_size.z) * step_size;

                if (tidx == 0) {
                        streamline[i] = point;
#if 0
                        if (threadIdx.y == 1) {
                                printf("streamline[%d]: %f, %f, %f\n", i, point.x, point.y, point.z);
                        }
#endif
                }
                __syncwarp(WMASK);

                tissue_class = check_point_d<BDIM_X, BDIM_Y>(tc_threshold, point, dimx, dimy, dimz, metric_map);

                if (tissue_class == ENDPOINT ||
                    tissue_class == INVALIDPOINT ||
                    tissue_class == OUTSIDEIMAGE) {
                        break;
                }
        }
        nsteps[0] = i;

        return tissue_class;
}

template<int BDIM_X,
         int BDIM_Y,
         typename REAL_T,
         typename REAL3_T>
__global__ void getNumStreamlines_k(const ModelType model_type,
                                    const REAL_T max_angle,
				    const REAL_T min_signal,
				    const REAL_T relative_peak_thres,
				    const REAL_T min_separation_angle,
				    const long long rndSeed,
                                    const int rndOffset,
                                    const int nseed,
                                    const REAL3_T *__restrict__ seeds,
                                    const int dimx,
                                    const int dimy,
                                    const int dimz,
                                    const int dimt,
                                    const REAL_T *__restrict__ dataf,
                                    const REAL_T *__restrict__ H,
                                    const REAL_T *__restrict__ R,
				    const int delta_nr,
                                    const REAL_T *__restrict__ delta_b,
                                    const REAL_T *__restrict__ delta_q,
                                    const int  *__restrict__ b0s_mask, // change to int
				    const int samplm_nr,
                                    const REAL_T *__restrict__ sampling_matrix,
                                    const REAL3_T *__restrict__ sphere_vertices,
                                    const int2 *__restrict__ sphere_edges,
                                    const int num_edges,
                                          REAL3_T *__restrict__ shDir0,
                                          int *slineOutOff) {

        const int tidx = threadIdx.x;
        const int slid = blockIdx.x*blockDim.y + threadIdx.y;
        const size_t gid = blockIdx.x * blockDim.y * blockDim.x + blockDim.x * threadIdx.y + threadIdx.x;

        if (slid >= nseed) {
                return;
        }

        REAL3_T seed = seeds[slid]; 
        // seed = lin_mat*seed + offset

        REAL3_T *__restrict__ __shDir = shDir0+slid*samplm_nr;

	// const int hr_side = dimt-1;

        hiprandStatePhilox4_32_10_t st;
        //hiprand_init(rndSeed, slid + rndOffset, DIV_UP(hr_side, BDIM_X)*tidx, &st); // each thread uses DIV_UP(hr_side/BDIM_X)
        hiprand_init(rndSeed, gid, 0, &st); // each thread uses DIV_UP(hr_side/BDIM_X)
                                                                                   // elements of the same sequence
        // python:
        //directions = get_direction(None, dataf, dwi_mask, sphere, s, H, R, model, max_angle,
        //                pmf_threshold, b0s_mask, min_signal, fit_matrix,
        //                sampling_matrix, init=True)

	//if (!tidx && !threadIdx.y && !blockIdx.x) {
	//	printf("seed: %f, %f, %f\n", seed.x, seed.y, seed.z);
	//}

        int ndir;
        if (model_type == PROB) {
                ndir = get_direction_prob_d<BDIM_X,
                                            BDIM_Y,
                                            1>(
                                                &st,
                                                dataf,
                                                max_angle,
                                                relative_peak_thres,
                                                min_separation_angle,
                                                MAKE_REAL3(0,0,0),
                                                dimx, dimy, dimz, dimt,
                                                seed,
                                                samplm_nr,
                                                sphere_vertices,
                                                sphere_edges,
                                                num_edges,
                                                __shDir);
        } else {
                ndir = get_direction_boot_d<BDIM_X,
                                            BDIM_Y,
                                            1>(
                                                &st,
                                                model_type,
                                                max_angle,
                                                min_signal,
                                                relative_peak_thres,
                                                min_separation_angle,
                                                MAKE_REAL3(0,0,0),
                                                dimx, dimy, dimz, dimt, dataf,
                                                b0s_mask /* !dwi_mask */,
                                                seed,
                                                H, R,
                                                // model unused
                                                // max_angle, pmf_threshold from global defines
                                                // b0s_mask already passed
                                                // min_signal from global defines
                                                delta_nr,
                                                delta_b, delta_q, // fit_matrix
                                                samplm_nr,
                                                sampling_matrix,
                                                sphere_vertices,
                                                sphere_edges,
                                                num_edges,
                                                __shDir);
        }
        if (tidx == 0) {
                slineOutOff[slid] = ndir;

		//if (!tidx && !threadIdx.y && !blockIdx.x) {
		//	printf("ndir: %d\n", ndir);
		//	for(int i = 0; i < ndir; i++) {
		//		printf("%f %f %f\n", __shDir[i].x, __shDir[i].y, __shDir[i].z);	
		//	}
		//}

        }

        return;
}

template<int BDIM_X,
         int BDIM_Y,
         typename REAL_T,
         typename REAL3_T>
__global__ void genStreamlinesMerge_k(const ModelType model_type,
				      const REAL_T max_angle,
				      const REAL_T min_signal,
				      const REAL_T tc_threshold,
				      const REAL_T step_size,
				      const REAL_T relative_peak_thres,
				      const REAL_T min_separation_angle,
				      const long long rndSeed,
                                      const int rndOffset,
                                      const int nseed,
                                      const REAL3_T *__restrict__ seeds,
                                      const int dimx,
                                      const int dimy,
                                      const int dimz,
                                      const int dimt,
                                      const REAL_T *__restrict__ dataf,
                                      const REAL_T *__restrict__ H,
                                      const REAL_T *__restrict__ R,
				      const int delta_nr,
                                      const REAL_T *__restrict__ delta_b,
                                      const REAL_T *__restrict__ delta_q,
                                      const int    *__restrict__ b0s_mask, // change to int
                                      const REAL_T *__restrict__ metric_map,
				      const int samplm_nr,
                                      const REAL_T *__restrict__ sampling_matrix,
                                      const REAL3_T *__restrict__ sphere_vertices,
                                      const int2 *__restrict__ sphere_edges,
                                      const int num_edges,
                                      const int    *__restrict__ slineOutOff,
                                            REAL3_T *__restrict__ shDir0,
                                            REAL3_T *__restrict__ shDir1,
                                            int     *__restrict__ slineSeed,
                                            int     *__restrict__ slineLen,
                                            REAL3_T *__restrict__ sline) {

        const int tidx = threadIdx.x;
        const int tidy = threadIdx.y;

        const int slid = blockIdx.x*blockDim.y + threadIdx.y;

        const int lid = (tidy*BDIM_X + tidx) % 32;
        const unsigned int WMASK = ((1ull << BDIM_X)-1) << (lid & (~(BDIM_X-1)));

	// const int hr_side = dimt-1;

        hiprandStatePhilox4_32_10_t st;
        // const int gbid = blockIdx.y*gridDim.x + blockIdx.x;
        const size_t gid = blockIdx.x * blockDim.y * blockDim.x + blockDim.x * threadIdx.y + threadIdx.x;
        //hiprand_init(rndSeed, slid+rndOffset, DIV_UP(hr_side, BDIM_X)*tidx, &st); // each thread uses DIV_UP(HR_SIDE/BDIM_X)
        hiprand_init(rndSeed, gid+1, 0, &st); // each thread uses DIV_UP(hr_side/BDIM_X)
                                                                                 // elements of the same sequence
        if (slid >= nseed) {
                return;
        }

        REAL3_T seed = seeds[slid]; 

        int ndir = slineOutOff[slid+1]-slineOutOff[slid];
#if 0
        if (threadIdx.y == 0 && threadIdx.x == 0) {
                printf("%s: ndir: %d\n", __func__, ndir);
                for(int i = 0; i < ndir; i++) {
                        printf("__shDir[%d][%d]: (%f, %f, %f)\n",
                                tidy, i, __shDir[tidy][i].x, __shDir[tidy][i].y, __shDir[tidy][i].z);
                }
        }
#endif
        __syncwarp(WMASK);

        int slineOff = slineOutOff[slid];

        for(int i = 0; i < ndir; i++) {

                REAL3_T first_step = shDir0[slid*samplm_nr + i];

		REAL3_T *__restrict__ currSline = sline + slineOff*MAX_SLINE_LEN*2;

                if (tidx == 0) {
                        slineSeed[slineOff] = slid;
                }
#if 0
                if (threadIdx.y == 0 && threadIdx.x == 0) {
                        printf("calling trackerF from: (%f, %f, %f)\n", first_step.x, first_step.y, first_step.z);
                }
#endif
                int stepsB;
                const int tissue_classB = tracker_d<BDIM_X,
                                                    BDIM_Y>(&st,
							    model_type,
						            max_angle,
						            min_signal,
							    tc_threshold,
							    step_size,
							    relative_peak_thres,
							    min_separation_angle,
                                                            seed,
                                                            MAKE_REAL3(-first_step.x, -first_step.y, -first_step.z),
                                                            MAKE_REAL3(1, 1, 1),
                                                            dimx, dimy, dimz, dimt, dataf,
                                                            b0s_mask,
                                                            H, R,
                                                            metric_map,
							    delta_nr,
                                                            delta_b, delta_q, //fit_matrix
							    samplm_nr,
                                                            sampling_matrix,
                                                            sphere_vertices,
                                                            sphere_edges,
                                                            num_edges,
                                                            shDir1 + slid*samplm_nr,
                                                            &stepsB,
                                                            currSline);
                //if (tidx == 0) {
                //        slineLenB[slineOff] = stepsB;
                //}

                // reverse backward sline
                for(int i = 0; i < stepsB/2; i += BDIM_X) {
                        if (i+tidx < stepsB/2) {
                                const REAL3_T __p = currSline[i+tidx];
                                currSline[i+tidx] = currSline[stepsB-1 - (i+tidx)];
                                currSline[stepsB-1 - (i+tidx)] = __p;
                        }
                }

                int stepsF;
                const int tissue_classF = tracker_d<BDIM_X,
                                                    BDIM_Y>(&st,
							    model_type,
						            max_angle,
						            min_signal,
							    tc_threshold,
							    step_size,
							    relative_peak_thres,
							    min_separation_angle,
                                                            seed,
                                                            first_step,
                                                            MAKE_REAL3(1, 1, 1),
                                                            dimx, dimy, dimz, dimt, dataf,
                                                            b0s_mask,
                                                            H, R,
                                                            metric_map,
							    delta_nr,
                                                            delta_b, delta_q, //fit_matrix
							    samplm_nr,
                                                            sampling_matrix,
                                                            sphere_vertices,
                                                            sphere_edges,
                                                            num_edges,
                                                            shDir1 + slid*samplm_nr,
                                                            &stepsF,
                                                            currSline + stepsB-1);
                if (tidx == 0) {
                        slineLen[slineOff] = stepsB-1+stepsF;
                }
                
                slineOff += 1;
#if 0
                if (threadIdx.y == 0 && threadIdx.x == 0) {
                        printf("%s: stepsF: %d, tissue_classF: %d\n", __func__, stepsF, tissue_classF);
                }
                __syncwarp(WMASK);
#endif
                //if (/* !return_all || */0 &&
                //    tissue_classF != ENDPOINT &&
                //    tissue_classF != OUTSIDEIMAGE) {
                //        continue;
                //}
                //if (/* !return_all || */ 0 &&
                //    tissue_classB != ENDPOINT &&
                //    tissue_classB != OUTSIDEIMAGE) {
                //        continue;
                //}
        }
        return;
}

void generate_streamlines_cuda_mgpu(const ModelType model_type, const REAL max_angle, const REAL min_signal, const REAL tc_threshold, const REAL step_size,
                                    const REAL relative_peak_thresh, const REAL min_separation_angle,
                                    const int nseeds, const std::vector<REAL*> &seeds_d,
                                    const int dimx, const int dimy, const int dimz, const int dimt,
                                    const std::vector<REAL*> &dataf_d, const std::vector<REAL*> &H_d, const std::vector<REAL*> &R_d,
			            const int delta_nr,
                                    const std::vector<REAL*> &delta_b_d, const std::vector<REAL*> &delta_q_d,
                                    const std::vector<int*> &b0s_mask_d, const std::vector<REAL*> &metric_map_d,
			            const int samplm_nr,
                                    const std::vector<REAL*> &sampling_matrix_d,
                                    const std::vector<REAL*> &sphere_vertices_d, const std::vector<int*> &sphere_edges_d, const int nedges,
                                    std::vector<REAL*> &slines_h, std::vector<int*> &slinesLen_h, std::vector<int> &nSlines_h,
                                    const std::vector<int> nSlines_old_h, const int rng_seed, const int rng_offset,
                                    const int ngpus, const std::vector<hipStream_t> &streams) {

  int nseeds_per_gpu = (nseeds + ngpus - 1) / ngpus;

  std::vector<int*> slinesOffs_d(ngpus, nullptr);
  std::vector<REAL3*> shDirTemp0_d(ngpus, nullptr);
  std::vector<REAL3*> shDirTemp1_d(ngpus, nullptr);

  //#pragma omp parallel for
  for (int n = 0; n < ngpus; ++n) {
    CHECK_CUDA(hipSetDevice(n));
    int nseeds_gpu = std::min(nseeds_per_gpu, std::max(0, nseeds - n*nseeds_per_gpu));
    dim3 block(THR_X_SL, THR_X_BL/THR_X_SL);
    dim3 grid(DIV_UP(nseeds_gpu, THR_X_BL/THR_X_SL));

    CHECK_CUDA(hipMalloc(&slinesOffs_d[n], sizeof(*slinesOffs_d[n])*(nseeds_gpu+1)));
    CHECK_CUDA(hipMalloc(&shDirTemp0_d[n], sizeof(*shDirTemp0_d[n])*samplm_nr*grid.x*block.y));
    CHECK_CUDA(hipMalloc(&shDirTemp1_d[n], sizeof(*shDirTemp1_d[n])*samplm_nr*grid.x*block.y));
  }

  int n32dimt = ((dimt+31)/32)*32;

  size_t shSizeGNS = sizeof(REAL)*(THR_X_BL/THR_X_SL)*(2*n32dimt + 2*MAX(n32dimt, samplm_nr)) + // for get_direction_boot_d
		     sizeof(int)*samplm_nr;						        // for peak_directions_d	

  //printf("shSizeGNS: %zu\n", shSizeGNS);

  //#pragma omp parallel for
  for (int n = 0; n < ngpus; ++n) {
    CHECK_CUDA(hipSetDevice(n));
    int nseeds_gpu = std::min(nseeds_per_gpu, std::max(0, nseeds - n*nseeds_per_gpu));
    if (nseeds_gpu == 0) continue;
    dim3 block(THR_X_SL, THR_X_BL/THR_X_SL);
    dim3 grid(DIV_UP(nseeds_gpu, THR_X_BL/THR_X_SL));

    // Precompute number of streamlines before allocating memory
    getNumStreamlines_k<THR_X_SL,
                        THR_X_BL/THR_X_SL>
                        <<<grid, block, shSizeGNS>>>(model_type,
                                                     max_angle,
						     min_signal,
						     relative_peak_thresh,
						     min_separation_angle,
						     rng_seed,
						     rng_offset + n*nseeds_per_gpu,
						     nseeds_gpu,
						     reinterpret_cast<const REAL3 *>(seeds_d[n]),
						     dimx,
						     dimy,
						     dimz,
						     dimt,
						     dataf_d[n],
						     H_d[n],
						     R_d[n],
						     delta_nr,
						     delta_b_d[n],
						     delta_q_d[n],
						     b0s_mask_d[n],
						     samplm_nr,
						     sampling_matrix_d[n],
						     reinterpret_cast<const REAL3 *>(sphere_vertices_d[n]),
						     reinterpret_cast<const int2 *>(sphere_edges_d[n]),
						     nedges,
						     shDirTemp0_d[n],
						     slinesOffs_d[n]);
  }

  std::vector<int> slinesOffs_h;
  //#pragma omp parallel for
  for (int n = 0; n < ngpus; ++n) {
    //std::vector<int> slinesOffs_h;
    int nseeds_gpu = std::min(nseeds_per_gpu, std::max(0, nseeds - n*nseeds_per_gpu));
    if (nseeds_gpu == 0) {
      nSlines_h[n] = 0;
      continue;
    }
    slinesOffs_h.resize(nseeds_gpu+1);
    CHECK_CUDA(hipMemcpy(slinesOffs_h.data(), slinesOffs_d[n], sizeof(*slinesOffs_h.data())*(nseeds_gpu+1), hipMemcpyDeviceToHost));

    int __pval = slinesOffs_h[0];
    slinesOffs_h[0] = 0;
    for(int i = 1; i < nseeds_gpu+1; i++) {
      const int __cval = slinesOffs_h[i];
      slinesOffs_h[i] = slinesOffs_h[i-1] + __pval;
      __pval = __cval;
    }
    nSlines_h[n] = slinesOffs_h[nseeds_gpu];
    CHECK_CUDA(hipMemcpy(slinesOffs_d[n], slinesOffs_h.data(), sizeof(*slinesOffs_d[n])*(nseeds_gpu+1), hipMemcpyHostToDevice));
  }

  std::vector<int*> slineSeed_d(ngpus, nullptr);

  //#pragma omp parallel for
  for (int n = 0; n < ngpus; ++n) {
    CHECK_CUDA(hipSetDevice(n));
    int nseeds_gpu = std::min(nseeds_per_gpu, std::max(0, nseeds - n*nseeds_per_gpu));

    CHECK_CUDA(hipMalloc(&slineSeed_d[n], sizeof(*slineSeed_d[n])*nSlines_h[n]));
    CHECK_CUDA(hipMemset(slineSeed_d[n], -1, sizeof(*slineSeed_d[n])*nSlines_h[n]));

    // Allocate/reallocate output arrays if necessary
    if (nSlines_h[n] > nSlines_old_h[n]) {
      if(slines_h[n]) hipHostFree(slines_h[n]);
      if(slinesLen_h[n]) hipHostFree(slinesLen_h[n]);
      slines_h[n] = nullptr;
      slinesLen_h[n] = nullptr;
    }

    if (!slines_h[n]) CHECK_CUDA(hipHostMalloc(&slines_h[n], 2*3*MAX_SLINE_LEN*nSlines_h[n]*sizeof(*slines_h[n])));
    if (!slinesLen_h[n]) CHECK_CUDA(hipHostMalloc(&slinesLen_h[n], nSlines_h[n]*sizeof(*slinesLen_h[n])));
  }

  //if (nSlines_h) {

  std::vector<int*> slineLen_d(ngpus, nullptr);
  std::vector<REAL3*> sline_d(ngpus, nullptr);
  //#pragma omp parallel for
  for (int n = 0; n < ngpus; ++n) {
    CHECK_CUDA(hipSetDevice(n));
    CHECK_CUDA(hipMalloc(&slineLen_d[n], sizeof(*slineLen_d[n])*nSlines_h[n]));

    CHECK_CUDA(hipMalloc(&sline_d[n], sizeof(*sline_d[n])*2*MAX_SLINE_LEN*nSlines_h[n]));

#if 0
    size_t free_mem, total_mem;
    CHECK_CUDA(hipMemGetInfo(&free_mem, &total_mem));
    std::cerr << "GPU " << n << ": ";
    std::cerr << "GPU Memory Usage before genStreamlinesMerge_k: ";
    std::cerr << (total_mem-free_mem)/(1024*1024) << " MiB used, ";
    std::cerr << total_mem/(1024*1024) << " MiB total ";
    std::cerr << std::endl;
#endif
  }

  //#pragma omp parallel for
  for (int n = 0; n < ngpus; ++n) {
    CHECK_CUDA(hipSetDevice(n));
    int nseeds_gpu = std::min(nseeds_per_gpu, std::max(0, nseeds - n*nseeds_per_gpu));
    if (nseeds_gpu == 0) continue;
    dim3 block(THR_X_SL, THR_X_BL/THR_X_SL);
    dim3 grid(DIV_UP(nseeds_gpu, THR_X_BL/THR_X_SL));
#if 0
    std::cerr << "GPU " << n << ": ";
    std::cerr << "Generating " << nSlines_h[n] << " streamlines (from " << nseeds_gpu << " seeds)" << std::endl; 
#endif
    //fprintf(stderr, "Launching kernel with %u blocks of size (%u, %u)\n", grid.x, block.x, block.y);
    genStreamlinesMerge_k<THR_X_SL,
                          THR_X_BL/THR_X_SL>
                          <<<grid, block, shSizeGNS, streams[n]>>>(model_type,
								   max_angle,
								   min_signal,
								   tc_threshold,
								   step_size,
								   relative_peak_thresh,
								   min_separation_angle,
								   rng_seed,
								   rng_offset + n*nseeds_per_gpu,
								   nseeds_gpu,
								   reinterpret_cast<const REAL3 *>(seeds_d[n]),
								   dimx,
								   dimy,
								   dimz,
								   dimt,
								   dataf_d[n],
								   H_d[n],
								   R_d[n],
								   delta_nr,
								   delta_b_d[n],
								   delta_q_d[n],
								   b0s_mask_d[n],
								   metric_map_d[n],
								   samplm_nr,
								   sampling_matrix_d[n],
								   reinterpret_cast<const REAL3 *>(sphere_vertices_d[n]),
								   reinterpret_cast<const int2 *>(sphere_edges_d[n]),
								   nedges,
								   slinesOffs_d[n],
								   shDirTemp0_d[n],
								   shDirTemp1_d[n],
								   slineSeed_d[n],
								   slineLen_d[n],
								   sline_d[n]);
    CHECK_ERROR("genStreamlinesMerge_k");
  }

  //CHECK_CUDA(hipDeviceSynchronize());

  //#pragma omp parallel for
  for (int n = 0; n < ngpus; ++n) {
    CHECK_CUDA(hipSetDevice(n));
    CHECK_CUDA(hipMemcpyAsync(slines_h[n],
                          reinterpret_cast<REAL *>(sline_d[n]),
                          sizeof(*slines_h[n])*2*MAX_SLINE_LEN*nSlines_h[n]*3,
                          hipMemcpyDeviceToHost, streams[n]));
    CHECK_CUDA(hipMemcpyAsync(slinesLen_h[n],
                          slineLen_d[n],
                          sizeof(*slinesLen_h[n])*nSlines_h[n],
                          hipMemcpyDeviceToHost, streams[n]));

  }
  //};

  //#pragma omp parallel for
  for (int n = 0; n < ngpus; ++n) {
    CHECK_CUDA(hipSetDevice(n));
    CHECK_CUDA(hipStreamSynchronize(streams[n]));
    CHECK_CUDA(hipFree(slineSeed_d[n]));
    CHECK_CUDA(hipFree(slinesOffs_d[n]));
    CHECK_CUDA(hipFree(shDirTemp0_d[n]));
    CHECK_CUDA(hipFree(shDirTemp1_d[n]));
    CHECK_CUDA(hipFree(slineLen_d[n]));
    CHECK_CUDA(hipFree(sline_d[n]));
  }

}

#if 1
void write_trk(const char *fname,
               const /*short*/ int *dims,
               const REAL *voxel_size,
               const char *voxel_order,
               const REAL *vox_to_ras,
               const int nsline,
               const int *slineLen,
               const REAL3 *sline) {

        FILE *fp = fopen(fname, "w");
        if (!fp) {
                fprintf(stderr, "Cannot open file %s for writing...\n", fname);
                exit(EXIT_FAILURE);
        }

        const char ID_STRING[6] = "TRACK";
        short DIM[3] = {1, 1, 1};
        float VOXEL_SIZE[3] = {1.0f, 1.0f, 1.0f};
        float VOX_TO_RAS[4][4] = {{1.0f, 0.0f, 0.0, 0.0f},
                                  {0.0f, 1.0f, 0.0, 0.0f},
                                  {0.0f, 0.0f, 1.0, 0.0f},
                                  {0.0f, 0.0f, 0.0, 1.0f}};
        //const char VOXEL_ORDER[2][4] = {"RAS", "LAS"};
        const float ORIGIN[3] = {0.0f, 0.0f, 0.0f};
        const float IMAGE_ORIENTATION_PATIENT[6] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
        const int VERSION = 2;
        const int HDR_SIZE = 1000;

        // write header
        unsigned char header[1000];
        memset(&header[0], 0, sizeof(header));

        long long int off = 0;

        memcpy(header, ID_STRING, sizeof(ID_STRING));
        off += sizeof(ID_STRING);

        if (dims) {
                DIM[0] = dims[0];
                DIM[1] = dims[1];
                DIM[2] = dims[2];
        }
        memcpy(header+off, DIM, sizeof(DIM));
        off += sizeof(DIM);

        if (voxel_size) {
                VOXEL_SIZE[0] = (float)voxel_size[0];
                VOXEL_SIZE[1] = (float)voxel_size[1];
                VOXEL_SIZE[2] = (float)voxel_size[2];
        }
        memcpy(header+off, VOXEL_SIZE, sizeof(VOXEL_SIZE));
        off += sizeof(VOXEL_SIZE);

        memcpy(header+off, ORIGIN, sizeof(ORIGIN));
        off += sizeof(ORIGIN);

        // skip n_scalaer(2b) + scalar_name(200b) +
        //      n_properties(2b) + property_name(200b)
        off += 404;

        if (vox_to_ras) {
                for(int i = 0; i < 4; i++) {
                        for(int j = 0; j < 4; j++) {
                                VOX_TO_RAS[i][j] = (float)vox_to_ras[i*4+j];
                        }
                }
        }
        memcpy(header+off, VOX_TO_RAS, sizeof(VOX_TO_RAS));
        off += sizeof(VOX_TO_RAS);

        // skip reserved(444b)
        off += 444;

        if (voxel_order) {
                memcpy(header+off, voxel_order, 4);
        } else {
                memcpy(header+off, "LAS", 4);
        }
        off += 4; //sizeof(VOXEL_ORDER[voxel_order]);

        // skip pad2(4b)
        off += 4;

        memcpy(header+off, IMAGE_ORIENTATION_PATIENT, sizeof(IMAGE_ORIENTATION_PATIENT));
        off += sizeof(IMAGE_ORIENTATION_PATIENT);

        // skip pad1(2b)
        off += 2;

        // skip invert_x(1b), invert_y(1b), invert_x(1b), swap_xy(1b), swap_yz(1b), swap_zx(1b)
        off += 6;

        memcpy(header+off, &nsline, sizeof(int));
        off += sizeof(int);

        memcpy(header+off, &VERSION, sizeof(VERSION));
        off += sizeof(VERSION);

        memcpy(header+off, &HDR_SIZE, sizeof(HDR_SIZE));
        off += sizeof(HDR_SIZE);

        //assert(off == 1000);
        if (off != 1000) {
                fprintf(stderr, "%s:%s:%d: heder size = %lld, (!= 1000)!\n", __FILE__, __func__, __LINE__, off);
                exit(EXIT_FAILURE);
        }
        
        size_t nw = fwrite(header, sizeof(header), 1, fp);
        if (nw != 1) {
                fprintf(stderr, "Error while writing to file!\n");
                exit(EXIT_FAILURE);
        }
#if 0
        // write body
        long long maxSlineLen = slineLen[0];
        for(long long i = 1; i < nsline; i++) {
                maxSlineLen = MAX(maxSlineLen, slineLen[i]);
        }

        float *slineData = (float *)Malloc((1+3*maxSlineLen)*sizeof(*slineData));
#else
        float slineData[1 + 3*(2*MAX_SLINE_LEN)];
#endif
        for(int i = 0; i < nsline; i++) {
                reinterpret_cast<int *>(slineData)[0] = slineLen[i];
                for(int j = 0; j < slineLen[i]; j++) {
                        slineData[1+3*j+0] = (float)((sline[i*2*MAX_SLINE_LEN + j].x+0.5)*VOXEL_SIZE[0]);
                        slineData[1+3*j+1] = (float)((sline[i*2*MAX_SLINE_LEN + j].y+0.5)*VOXEL_SIZE[1]);
                        slineData[1+3*j+2] = (float)((sline[i*2*MAX_SLINE_LEN + j].z+0.5)*VOXEL_SIZE[2]);
                }
                nw = fwrite(slineData, (1+3*slineLen[i])*sizeof(*slineData), 1, fp);
                if (nw != 1) {
                        fprintf(stderr, "Error while writing to file!\n");
                        exit(EXIT_FAILURE);
                }
        }
#if 0
        free(slineData);
#endif
        fclose(fp);

        return;
}
#else
void write_trk(const int num_threads,
               const char *fname,
               const /*short*/ int *dims,
               const REAL *voxel_size,
               const char *voxel_order,
               const REAL *vox_to_ras,
               const int nsline,
               const int *slineLen,
               const REAL3 *sline) {

        FILE *fp = fopen(fname, "w");
        if (!fp) {
                fprintf(stderr, "Cannot open file %s for writing...\n", fname);
                exit(EXIT_FAILURE);
        }

        const char ID_STRING[6] = "TRACK";
        short DIM[3] = {1, 1, 1};
        float VOXEL_SIZE[3] = {1.0f, 1.0f, 1.0f};
        float VOX_TO_RAS[4][4] = {{1.0f, 0.0f, 0.0, 0.0f},
                                  {0.0f, 1.0f, 0.0, 0.0f},
                                  {0.0f, 0.0f, 1.0, 0.0f},
                                  {0.0f, 0.0f, 0.0, 1.0f}};
        //const char VOXEL_ORDER[2][4] = {"RAS", "LAS"};
        const float ORIGIN[3] = {0.0f, 0.0f, 0.0f};
        const float IMAGE_ORIENTATION_PATIENT[6] = {0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f};
        const int VERSION = 2;
        const int HDR_SIZE = 1000;

        // write header
        unsigned char header[1000];
        memset(&header[0], 0, sizeof(header));

        long long int off = 0;

        memcpy(header, ID_STRING, sizeof(ID_STRING));
        off += sizeof(ID_STRING);

        if (dims) {
                DIM[0] = dims[0];
                DIM[1] = dims[1];
                DIM[2] = dims[2];
        }
        memcpy(header+off, DIM, sizeof(DIM));
        off += sizeof(DIM);

        if (voxel_size) {
                VOXEL_SIZE[0] = (float)voxel_size[0];
                VOXEL_SIZE[1] = (float)voxel_size[1];
                VOXEL_SIZE[2] = (float)voxel_size[2];
        }
        memcpy(header+off, VOXEL_SIZE, sizeof(VOXEL_SIZE));
        off += sizeof(VOXEL_SIZE);

        memcpy(header+off, ORIGIN, sizeof(ORIGIN));
        off += sizeof(ORIGIN);

        // skip n_scalaer(2b) + scalar_name(200b) +
        //      n_properties(2b) + property_name(200b)
        off += 404;

        if (vox_to_ras) {
                for(int i = 0; i < 4; i++) {
                        for(int j = 0; j < 4; j++) {
                                VOX_TO_RAS[i][j] = (float)vox_to_ras[i*4+j];
                        }
                }
        }
        memcpy(header+off, VOX_TO_RAS, sizeof(VOX_TO_RAS));
        off += sizeof(VOX_TO_RAS);

        // skip reserved(444b)
        off += 444;

        if (voxel_order) {
                memcpy(header+off, voxel_order, 4);
        } else {
                memcpy(header+off, "LAS", 4);
        }
        off += 4; //sizeof(VOXEL_ORDER[voxel_order]);

        // skip pad2(4b)
        off += 4;

        memcpy(header+off, IMAGE_ORIENTATION_PATIENT, sizeof(IMAGE_ORIENTATION_PATIENT));
        off += sizeof(IMAGE_ORIENTATION_PATIENT);

        // skip pad1(2b)
        off += 2;

        // skip invert_x(1b), invert_y(1b), invert_x(1b), swap_xy(1b), swap_yz(1b), swap_zx(1b)
        off += 6;

        memcpy(header+off, &nsline, sizeof(int));
        off += sizeof(int);

        memcpy(header+off, &VERSION, sizeof(VERSION));
        off += sizeof(VERSION);

        memcpy(header+off, &HDR_SIZE, sizeof(HDR_SIZE));
        off += sizeof(HDR_SIZE);

        //assert(off == 1000);
        if (off != 1000) {
                fprintf(stderr, "%s:%s:%d: heder size = %lld, (!= 1000)!\n", __FILE__, __func__, __LINE__, off);
                exit(EXIT_FAILURE);
        }
        
        size_t nw = fwrite(header, sizeof(header), 1, fp);
        if (nw != 1) {
                fprintf(stderr, "Error while writing to file!\n");
                exit(EXIT_FAILURE);
        }

        // write body
        long long maxSlineLen = slineLen[0];
        for(long long i = 1; i < nsline; i++) {
                maxSlineLen = MAX(maxSlineLen, slineLen[i]);
        }

        //omp_set_dynamic(0);
        const int NTHREADS = num_threads > 0 ? num_threads : 1;
        omp_set_num_threads(NTHREADS);

        const int NFLTS_PER_TH = 1 + 2*(3*MAX_SLINE_LEN);
        float *slineData = (float *)Malloc(NFLTS_PER_TH*NTHREADS*sizeof(*slineData));

        #pragma omp parallel 
        {
                const int tid = omp_get_thread_num();
                float *__mySlineData = slineData+tid*NFLTS_PER_TH;
#if 1
                //#pragma omp for schedule(static)
                for(int i = 0; i < nsline; i += NTHREADS) {
                        if (i+tid < nsline) {
                                reinterpret_cast<int *>(__mySlineData)[0] = slineLen[i+tid];
                                for(int j = 0; j < slineLen[i+tid]; j++) {
                                        __mySlineData[1+3*j+0] = (float)((sline[(i+tid)*2*MAX_SLINE_LEN + j].x+0.5)*VOXEL_SIZE[0]);
                                        __mySlineData[1+3*j+1] = (float)((sline[(i+tid)*2*MAX_SLINE_LEN + j].y+0.5)*VOXEL_SIZE[1]);
                                        __mySlineData[1+3*j+2] = (float)((sline[(i+tid)*2*MAX_SLINE_LEN + j].z+0.5)*VOXEL_SIZE[2]);
                                }
                        }
                        #pragma omp barrier
                        if (tid == 0) {
                                for(int j = 0; j < NTHREADS; j++) {
                                        if (i+j >= nsline) {
                                               break;
                                        }
                                        nw = fwrite(slineData+j*NFLTS_PER_TH, (1+3*slineLen[i+j])*sizeof(*slineData), 1, fp);
                                        if (nw != 1) {
                                                fprintf(stderr, "Error while writing to file!\n");
                                                exit(EXIT_FAILURE);
                                        }
                                }
                        }
                        #pragma omp barrier
                }
#else
                // streamlines are not required to be in any specific order inside the trk file...
                #pragma omp for
                for(int i = 0; i < nsline; i++) {
                        reinterpret_cast<int *>(__mySlineData)[0] = slineLen[i];
                        for(int j = 0; j < slineLen[i]; j++) {
                                __mySlineData[1+3*j+0] = (float)((sline[i*2*MAX_SLINE_LEN + j].x+0.5)*VOXEL_SIZE[0]);
                                __mySlineData[1+3*j+1] = (float)((sline[i*2*MAX_SLINE_LEN + j].y+0.5)*VOXEL_SIZE[1]);
                                __mySlineData[1+3*j+2] = (float)((sline[i*2*MAX_SLINE_LEN + j].z+0.5)*VOXEL_SIZE[2]);
                        }
                        nw = fwrite(__mySlineData, (1+3*slineLen[i])*sizeof(*__mySlineData), 1, fp);
                        if (nw != 1) {
                                fprintf(stderr, "Error while writing to file!\n");
                                exit(EXIT_FAILURE);
                        }
                }
#endif
        }
        free(slineData);
        fclose(fp);

        return;
}
#endif
